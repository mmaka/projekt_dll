#include "hip/hip_runtime.h"
#include"CudaTekstury.cuh"

__constant__ size rozmiaryDanych[6];

void CudaTekstury::init() {

	hipDeviceProp_t prop;
	int dev;
	memset(&prop, 0, sizeof(hipDeviceProp_t));
	prop.major = 1;
	prop.minor = 0;
	HANDLE_ERROR(hipChooseDevice(&dev, &prop));
	HANDLE_ERROR(cudaGLSetGLDevice(dev));
	
	auto f = std::async(std::launch::async, [&] {

		ustawMapeKolorow();
		HANDLE_ERROR(hipMalloc(&d_mapaKolory_Szarosc, 256 * sizeof(uchar4)));
		HANDLE_ERROR(hipMemcpy(d_mapaKolory_Szarosc, mapaKolorySzarosc, 256 * sizeof(uchar4), hipMemcpyHostToDevice));

	});

	streams.resize(liczbaStrumieni);
	for (size_t i = 0; i < liczbaStrumieni; ++i)
		HANDLE_ERROR(hipStreamCreate(&streams[i]));

	inicjalizacja = true;
	
}

void CudaTekstury::pobierzDaneCPU() {
	LARGE_INTEGER countPerSec, tim1, tim2;
	QueryPerformanceFrequency(&countPerSec);
	QueryPerformanceCounter(&tim1);
	HANDLE_ERROR(hipMalloc(&daneGPU, calkowityRozmiarDanych() *sizeof(oct_t)));
	HANDLE_ERROR(hipMemcpy(daneGPU, daneCPU, calkowityRozmiarDanych() * sizeof(oct_t), hipMemcpyHostToDevice));
	QueryPerformanceCounter(&tim2);
	double j = (double)(tim2.QuadPart - tim1.QuadPart) / countPerSec.QuadPart * 1000;
	printf("czas kopiowania: %f\n", j);
	delete[] daneCPU;
	daneCPU = nullptr;
}

void CudaTekstury::ustawMapeKolorow() {

	int progCzerni = jasnosc - (kontrast / 2);//do progu czerni kolor czarny
	

	for (int i = 0, end = (progCzerni<256) ? progCzerni : 256; i != end; ++i) {
		
		mapaKolorySzarosc[i].x = 0;
		mapaKolorySzarosc[i].y = 0;
		mapaKolorySzarosc[i].z = 0;
		mapaKolorySzarosc[i].w = 0;
		
	}
	//miedzy czarnym i bia�ym przedzia� kolorow
	int progSzarosciKolorow = jasnosc + (kontrast / 2);
	//tutaj jest b��d: progCzerni moze byc wieksz niz 256 -> spojrz na warunek konca powy�szej p�tli
	for (int i = progCzerni, przedzial = progSzarosciKolorow - progCzerni, end = (progSzarosciKolorow<256) ? progSzarosciKolorow : 256; i != end; ++i) {

		unsigned char val = (unsigned char)((255 * ((float)i - progCzerni)) / przedzial);
		mapaKolorySzarosc[i].x = defKol[val][0];
		mapaKolorySzarosc[i].y = defKol[val][1];
		mapaKolorySzarosc[i].z = defKol[val][2];
		mapaKolorySzarosc[i].w = val;

	}
	//powyzej progu szarosci kolor bialy
	for (int i = progSzarosciKolorow; i != 256; ++i) {

		mapaKolorySzarosc[i].x = 255;
		mapaKolorySzarosc[i].y = 255;
		mapaKolorySzarosc[i].z = 255;
		mapaKolorySzarosc[i].w = 255;

	}
}


void CudaTekstury::edycjaMapyKolorow(EDYCJA_MAPY_KOLOROW tryb, int value) {

	//gwarantujemy, �e kolejne przetwarzanie mapy zacznie si� po zako�czeniu trwaj�cego przetwarzania
	if (!przetwarzanieMapyKolorow.test_and_set()) {
		
		//przed przetworzeniem mapy kolor�w i uruchomieniem funkcji j�dra sprawdzamy czy warto�ci kontrastu i jasno�ci nale�� do odpowiedniego zakresu
		//np. gdy kontrast osi�gnie warto�� 0, zmniejszanie warto�ci nie powinno mie� miejsca
		bool czyOdswiezamyKoloryPrzekrojow = false;

		switch (tryb)
		{
		case EDYCJA_MAPY_KOLOROW::ZWIEKSZ_KONTRAST:
		{
			int tmp = kontrast - value;

			if (tmp >= 0) {

				kontrast = tmp;
				czyOdswiezamyKoloryPrzekrojow = true;

			}
			else {

				kontrast = 0;

			}
		}
		break;
		case EDYCJA_MAPY_KOLOROW::ZMNIEJSZ_KONTRAST:
		{
			int tmp = kontrast + value;

			if (tmp <= 256) {

				kontrast = tmp;
				czyOdswiezamyKoloryPrzekrojow = true;

			}
			else {

				kontrast = 256;
			}
		}
		break;
		case EDYCJA_MAPY_KOLOROW::ZWIEKSZ_JASNOSC:
		{
			int tmp = jasnosc - value;

			if (tmp >= 0) {

				jasnosc = tmp;
				czyOdswiezamyKoloryPrzekrojow = true;

			}
			else {

				jasnosc = 0;
			}
		}
		break;
		case EDYCJA_MAPY_KOLOROW::ZMNIEJSZ_JASNOSC:
		{
			int tmp = jasnosc + value;

			if (jasnosc <= 256) {

				jasnosc = tmp;
				czyOdswiezamyKoloryPrzekrojow = true;

			}
			else {

				jasnosc = 256;
			}
		}
		break;
		default:
			break;
		}

		if (czyOdswiezamyKoloryPrzekrojow) {

			ustawMapeKolorow();
			LARGE_INTEGER countPerSec, tim1, tim2;
			QueryPerformanceFrequency(&countPerSec);
			QueryPerformanceCounter(&tim1);
			HANDLE_ERROR(hipMemcpy(d_mapaKolory_Szarosc, mapaKolorySzarosc, 256 * sizeof(uchar4), hipMemcpyHostToDevice));
			QueryPerformanceCounter(&tim2);
			double j = (double)(tim2.QuadPart - tim1.QuadPart) / countPerSec.QuadPart * 1000;
			//	printf("czas kopiowania mapy: %f\n", j);
			pokolorujTeksturyIprzeslijDoTablicCuda();

		}

		przetwarzanieMapyKolorow.clear();
	}
	
}
void CudaTekstury::trybWyswietlaniaRGBczyGS(){

	if (!zmianaTrybuRGBnaGS.test_and_set()) {

		kolor = !kolor;
		pokolorujTeksturyIprzeslijDoTablicCuda();
		zmianaTrybuRGBnaGS.clear();

	}
}


void CudaTekstury::sprzatanie() {

	if (inicjalizacja) {

		HANDLE_ERROR(hipFree(d_mapaKolory_Szarosc));
	//	HANDLE_ERROR(hipFree(daneGPU));
	//	HANDLE_ERROR(hipFree(daneGPU_bskan_oct));
	//	HANDLE_ERROR(hipFree(daneGPU_ppop_oct));
	//	HANDLE_ERROR(hipFree(daneGPU_ppoz_oct));

		HANDLE_ERROR(hipFree(daneGPU_bskan_kolor));
		if (trybWyswietlania == WIZUALIZACJA::TYP_3D) {

			HANDLE_ERROR(hipFree(daneGPU_ppop_kolor));
			HANDLE_ERROR(hipFree(daneGPU_ppoz_kolor));
		}
		
		for (size_t i = 0; i < liczbaStrumieni; ++i) HANDLE_ERROR(hipStreamDestroy(streams[i]));

	}
	
	//if (daneGPU != nullptr) delete[] daneGPU;
	if (daneCPU != nullptr) delete[] daneCPU;

}

__global__ void przepisanieObuPrzekrojow(const uchar4 *dstGPU, uchar4 * dstGPU_ppop, uchar4 * dstGPU_ppoz) {

	if (threadIdx.x < blockDim.x) {

		dstGPU_ppop[blockIdx.x*blockDim.x*gridDim.y + blockIdx.y*blockDim.x + threadIdx.x] = dstGPU[blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x]; 
		dstGPU_ppoz[threadIdx.x*gridDim.y*gridDim.x + blockIdx.x*gridDim.y + blockIdx.y] = dstGPU[blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x];
																								
	}
}

__global__ void kolorowanie_bskan(uchar4 *dstGPU, const oct_t *source, const uchar4 *kolory ) {

	if (threadIdx.x < blockDim.x) {

		dstGPU[blockIdx.x*blockDim.x + threadIdx.x] = kolory[(unsigned char)source[blockIdx.x*blockDim.x + threadIdx.x]];
		dstGPU[blockIdx.x*blockDim.x + threadIdx.x].w = (unsigned char)source[blockIdx.x*blockDim.x + threadIdx.x];	
	}
}

__global__ void kolorowanie_ppop(uchar4 *dstGPU, const oct_t *source, const uchar4 *kolory) {

	if (threadIdx.x < blockDim.x) {

		dstGPU[blockIdx.x*blockDim.x + threadIdx.x] = kolory[(unsigned char)source[blockIdx.x*blockDim.x + threadIdx.x]];
		dstGPU[blockIdx.x*blockDim.x + threadIdx.x].w = (unsigned char)source[blockIdx.x*blockDim.x + threadIdx.x];
	}
}

__global__ void kolorowanie_ppoz(uchar4 *dstGPU, const oct_t *source, const uchar4 *kolory) {

	if (threadIdx.x < blockDim.x) {

		dstGPU[blockIdx.x*blockDim.x + threadIdx.x] = kolory[(unsigned char)source[blockIdx.x*blockDim.x + threadIdx.x]];
		dstGPU[blockIdx.x*blockDim.x + threadIdx.x].w = (unsigned char)source[blockIdx.x*blockDim.x + threadIdx.x];
	}
}

__global__ void wybraniePpoz(const oct_t *dstGPU, oct_t * dstGPU_ppoz,size krok_ppoz,size szerB) {

	if (threadIdx.x < blockDim.x) {

	//	register int ktory_ppoz = blockIdx.y*krok_ppoz;//chyba bez sensu
		dstGPU_ppoz[blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x] = dstGPU[threadIdx.x*gridDim.x*szerB + blockIdx.x*szerB + blockIdx.y*krok_ppoz];
	//	dstGPU_ppoz[blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x] = dstGPU[threadIdx.x*gridDim.x*rozmiaryDanych[0] + blockIdx.x*rozmiaryDanych[0] + blockIdx.y*rozmiaryDanych[5]];
	}
}


__global__ void wybraniePpop(const oct_t *dstGPU, oct_t * dstGPU_ppop, size krok_ppop, size rozA) {

	if (threadIdx.x < blockDim.x) {

	//	register int ktory_ppop = blockIdx.x * krok_ppop;
		dstGPU_ppop[blockIdx.x*blockDim.x*gridDim.y + blockIdx.y*blockDim.x + threadIdx.x] = dstGPU[blockIdx.y*rozA * blockDim.x + blockIdx.x * krok_ppop * blockDim.x + threadIdx.x];
		//dstGPU_ppop[blockIdx.x*blockDim.x*gridDim.y + blockIdx.y*blockDim.x + threadIdx.x] = dstGPU[blockIdx.y*rozmiaryDanych[1] *blockDim.x + blockIdx.x * rozmiaryDanych[4] *blockDim.x + threadIdx.x];

	}
}


__global__ void wybranieBskanow(const oct_t *dstGPU, oct_t * dstGPU_bskan,size krok_bskany) {

	if (threadIdx.x < blockDim.x) {

		//register int ktory_bskan = blockIdx.y*krok_bskany;
		//dstGPU_bskan[blockIdx.y*blockDim.x*gridDim.x + blockIdx.x*blockDim.x + threadIdx.x] = dstGPU[blockIdx.y*rozmiaryDanych[3]*blockDim.x*gridDim.x + blockIdx.x*blockDim.x + threadIdx.x];
		dstGPU_bskan[blockIdx.y*blockDim.x*gridDim.x + blockIdx.x*blockDim.x + threadIdx.x] = dstGPU[blockIdx.y*krok_bskany * blockDim.x*gridDim.x + blockIdx.x*blockDim.x + threadIdx.x];

	}
}

__global__ void wyborIKolorowaniePpoz(const oct_t *dstGPU, uchar4 * dstGPU_ppoz, size krok_ppoz, size szerB, const uchar4* kolory) {

	if (threadIdx.x < blockDim.x) {

		register int ktory_ppoz = blockIdx.y*krok_ppoz;//chyba bez sensu
		dstGPU_ppoz[blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x] = kolory[dstGPU[threadIdx.x*gridDim.x*szerB + blockIdx.x*szerB + ktory_ppoz]];
	}
}


__global__ void  wyborIKolorowaniePpop(const oct_t *dstGPU, uchar4 * dstGPU_ppop, size krok_ppop, size rozA, const uchar4* kolory) {

	if (threadIdx.x < blockDim.x) {

		register int ktory_ppop = blockIdx.x * krok_ppop;
		dstGPU_ppop[blockIdx.x*blockDim.x*gridDim.y + blockIdx.y*blockDim.x + threadIdx.x] = kolory[dstGPU[blockIdx.y*rozA*blockDim.x + ktory_ppop*blockDim.x + threadIdx.x]]; //dstGPU[nrB + nrW + nrKol];

	}
}


__global__ void  wyborIKolorowanieBskan(const oct_t *dstGPU, uchar4 * dstGPU_bskan, size krok_bskany,const uchar4* kolory) {

	if (threadIdx.x < blockDim.x) {

		register int ktory_bskan = blockIdx.y*krok_bskany;
		dstGPU_bskan[blockIdx.y*blockDim.x*gridDim.x + blockIdx.x*blockDim.x + threadIdx.x] = kolory[dstGPU[ktory_bskan*blockDim.x*gridDim.x + blockIdx.x*blockDim.x + threadIdx.x]];
	}
}


void CudaTekstury::kolorowanieBskan() {

	HANDLE_ERROR(hipMalloc(&daneGPU_bskan_kolor, szerokoscBskanu*rozmiarAskanu*liczbaBskanow * sizeof(uchar4)));
	dim3 block(szerokoscBskanu);
	dim3 grid(rozmiarAskanu*liczbaBskanow);

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(kolorowanie_bskan), hipFuncCachePreferL1);
	LARGE_INTEGER countPerSec, tim1, tim2;
	QueryPerformanceFrequency(&countPerSec);
	QueryPerformanceCounter(&tim1);
	kolorowanie_bskan << <grid, block, 0, streams[0] >> >(daneGPU_bskan_kolor, daneGPU_bskan_oct, d_mapaKolory_Szarosc);
	//kolorowanie_bskan << <grid, block>> >(daneGPU_bskan_kolor, daneGPU_bskan_oct, d_mapaKolory_Szarosc);
	QueryPerformanceCounter(&tim2);
	double j = (double)(tim2.QuadPart - tim1.QuadPart) / countPerSec.QuadPart * 1000;
	//	printf("czas kolorowania: %f\n", j);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernelll launch failed: %s\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);
	}
}

void CudaTekstury::kolorowaniePpop() {

	HANDLE_ERROR(hipMalloc(&daneGPU_ppop_kolor, liczbaPrzekrojowPoprzecznych*szerokoscBskanu*glebokoscPomiaru * sizeof(uchar4)));

	dim3 block(szerokoscBskanu);
	dim3 grid(liczbaPrzekrojowPoprzecznych*glebokoscPomiaru);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(kolorowanie_ppop), hipFuncCachePreferL1);
	LARGE_INTEGER countPerSec, tim1, tim2;
	QueryPerformanceFrequency(&countPerSec);
	QueryPerformanceCounter(&tim1);
	kolorowanie_ppop << <grid, block, 0, streams[1] >> >(daneGPU_ppop_kolor, daneGPU_ppop_oct, d_mapaKolory_Szarosc);
	//kolorowanie_ppop << <grid, block>> >(daneGPU_ppop_kolor, daneGPU_ppop_oct, d_mapaKolory_Szarosc);
	QueryPerformanceCounter(&tim2);
	double j = (double)(tim2.QuadPart - tim1.QuadPart) / countPerSec.QuadPart * 1000;
	//	printf("czas kolorowania: %f\n", j);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernelll launch failed: %s\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);
	}
}

void CudaTekstury::kolorowaniePpoz() {

	HANDLE_ERROR(hipMalloc(&daneGPU_ppoz_kolor, rozmiarAskanu*glebokoscPomiaru*liczbaPrzekrojowPoziomych * sizeof(uchar4)));

	dim3 block(glebokoscPomiaru);
	dim3 grid(rozmiarAskanu*liczbaPrzekrojowPoziomych);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(kolorowanie_ppoz), hipFuncCachePreferL1);
	LARGE_INTEGER countPerSec, tim1, tim2;
	QueryPerformanceFrequency(&countPerSec);
	QueryPerformanceCounter(&tim1);
	kolorowanie_ppoz << <grid, block, 0, streams[2] >> >(daneGPU_ppoz_kolor, daneGPU_ppoz_oct, d_mapaKolory_Szarosc);	
	//kolorowanie_ppoz << <grid, block>> >(daneGPU_ppoz_kolor, daneGPU_ppoz_oct, d_mapaKolory_Szarosc);
	QueryPerformanceCounter(&tim2);
	double j = (double)(tim2.QuadPart - tim1.QuadPart) / countPerSec.QuadPart * 1000;
	//	printf("czas kolorowania: %f\n", j);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernelll launch failed: %s\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);
	}
}

void CudaTekstury::ppoz_przepisanie_i_kolorowanie() {


	HANDLE_ERROR(hipMalloc(&daneGPU_ppoz_kolor, rozmiarAskanu*glebokoscPomiaru*liczbaPrzekrojowPoziomych * sizeof(uchar4)));

	dim3 block(glebokoscPomiaru);
	dim3 grid(rozmiarAskanu, liczbaPrzekrojowPoziomych);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(wyborIKolorowaniePpoz), hipFuncCachePreferL1);
	LARGE_INTEGER countPerSec, tim1, tim2;
	QueryPerformanceFrequency(&countPerSec);
	QueryPerformanceCounter(&tim1);
	//wyborIKolorowaniePpoz << <grid, block>> > (daneGPU, daneGPU_ppoz_kolor, krok_przekrojePoziome, szerokoscBskanu, mapaKolorySzarosc);
	wyborIKolorowaniePpoz << <grid, block, 0, streams[2] >> > (daneGPU, daneGPU_ppoz_kolor, krok_przekrojePoziome, szerokoscBskanu, mapaKolorySzarosc);
	QueryPerformanceCounter(&tim2);
	double j = (double)(tim2.QuadPart - tim1.QuadPart) / countPerSec.QuadPart * 1000;
	//	printf("czas przepisania: %f\n", j);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernellll launch failed: %s\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);
	}
}


void CudaTekstury::ppop_przepisanie_i_kolorowanie() {


	HANDLE_ERROR(hipMalloc(&daneGPU_ppop_kolor, szerokoscBskanu*glebokoscPomiaru*liczbaPrzekrojowPoprzecznych * sizeof(uchar4)));

	dim3 block(szerokoscBskanu);
	dim3 grid(liczbaPrzekrojowPoprzecznych, glebokoscPomiaru);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(wyborIKolorowaniePpop), hipFuncCachePreferL1);
	LARGE_INTEGER countPerSec, tim1, tim2;
	QueryPerformanceFrequency(&countPerSec);
	QueryPerformanceCounter(&tim1);
	wyborIKolorowaniePpop << <grid, block, 0, streams[1] >> > (daneGPU, daneGPU_ppop_kolor, krok_przekrojePoprzeczne, rozmiarAskanu, mapaKolorySzarosc);
	//wyborIKolorowaniePpop << <grid, block>> > (daneGPU, daneGPU_ppop_kolor, krok_przekrojePoprzeczne, rozmiarAskanu, mapaKolorySzarosc);
	QueryPerformanceCounter(&tim2);
	double j = (double)(tim2.QuadPart - tim1.QuadPart) / countPerSec.QuadPart * 1000;
	//	printf("czas przepisania: %f\n", j);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernellll launch failed: %s\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);
	}
}


void CudaTekstury::bskan_przepisanie_i_kolorowanie() {


	HANDLE_ERROR(hipMalloc(&daneGPU_bskan_kolor, szerokoscBskanu*liczbaBskanow*rozmiarAskanu * sizeof(uchar4)));

	dim3 block(szerokoscBskanu);
	dim3 grid(rozmiarAskanu, liczbaBskanow);

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(wyborIKolorowanieBskan), hipFuncCachePreferL1);
	LARGE_INTEGER countPerSec, tim1, tim2;
	QueryPerformanceFrequency(&countPerSec);
	QueryPerformanceCounter(&tim1);
	wyborIKolorowanieBskan << <grid, block, 0, streams[0] >> > (daneGPU, daneGPU_bskan_kolor, krok_bskan,mapaKolorySzarosc);
	//wyborIKolorowanieBskan << <grid, block>> > (daneGPU, daneGPU_bskan_kolor, krok_bskan, mapaKolorySzarosc);
	QueryPerformanceCounter(&tim2);
	double j = (double)(tim2.QuadPart - tim1.QuadPart) / countPerSec.QuadPart * 1000;
	//	printf("czas przepisania: %f\n", j);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernellll launch failed: %s\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);
	}
}


void CudaTekstury::przepisanie_oct_t_ppoz() {

	HANDLE_ERROR(hipMalloc(&daneGPU_ppoz_oct, rozmiarAskanu*glebokoscPomiaru*liczbaPrzekrojowPoziomych * sizeof(oct_t)));

	dim3 block(glebokoscPomiaru);
	dim3 grid(rozmiarAskanu, liczbaPrzekrojowPoziomych);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(wybraniePpoz), hipFuncCachePreferL1);
	LARGE_INTEGER countPerSec, tim1, tim2;
	QueryPerformanceFrequency(&countPerSec);
	QueryPerformanceCounter(&tim1);
	wybraniePpoz << <grid, block, 0, streams[2] >> > (daneGPU,daneGPU_ppoz_oct,krok_przekrojePoziome,szerokoscBskanu);
	//wybraniePpoz << <grid, block>> > (daneGPU, daneGPU_ppoz_oct, krok_przekrojePoziome, szerokoscBskanu);
	QueryPerformanceCounter(&tim2);
	double j = (double)(tim2.QuadPart - tim1.QuadPart) / countPerSec.QuadPart * 1000;
	//	printf("czas przepisania: %f\n", j);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernellll launch failed: %s\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);
	}
}


void CudaTekstury::przepisanie_oct_t_ppop() {


	HANDLE_ERROR(hipMalloc(&daneGPU_ppop_oct, szerokoscBskanu*glebokoscPomiaru*liczbaPrzekrojowPoprzecznych * sizeof(oct_t)));
	
	dim3 block(szerokoscBskanu);
	dim3 grid(liczbaPrzekrojowPoprzecznych, glebokoscPomiaru);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(wybraniePpop), hipFuncCachePreferL1);
	LARGE_INTEGER countPerSec, tim1, tim2;
	QueryPerformanceFrequency(&countPerSec);
	QueryPerformanceCounter(&tim1);
	wybraniePpop << <grid, block, 0, streams[1] >> > (daneGPU, daneGPU_ppop_oct,krok_przekrojePoprzeczne,rozmiarAskanu);
	//wybraniePpop << <grid, block>> > (daneGPU, daneGPU_ppop_oct, krok_przekrojePoprzeczne, rozmiarAskanu);
	QueryPerformanceCounter(&tim2);
	double j = (double)(tim2.QuadPart - tim1.QuadPart) / countPerSec.QuadPart * 1000;
	//	printf("czas przepisania: %f\n", j);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernellll launch failed: %s\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);
	}
}


void CudaTekstury::przepisanie_oct_t_bskan() {


	HANDLE_ERROR(hipMalloc(&daneGPU_bskan_oct, szerokoscBskanu*liczbaBskanow*rozmiarAskanu * sizeof(oct_t)));

	dim3 block(szerokoscBskanu);
	dim3 grid(rozmiarAskanu, liczbaBskanow);
	
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(wybranieBskanow), hipFuncCachePreferL1);
	LARGE_INTEGER countPerSec, tim1, tim2;
	QueryPerformanceFrequency(&countPerSec);
	QueryPerformanceCounter(&tim1);
	wybranieBskanow << <grid, block,0,streams[0] >> > (daneGPU, daneGPU_bskan_oct,krok_bskan);
	//wybranieBskanow << <grid, block>> > (daneGPU, daneGPU_bskan_oct, krok_bskan);
	QueryPerformanceCounter(&tim2);
	double j = (double)(tim2.QuadPart - tim1.QuadPart) / countPerSec.QuadPart * 1000;
	//	printf("czas przepisania: %f\n", j);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernellll launch failed: %s\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);
	}

}

void CudaTekstury::przepisanie_oct_t() {

	bskan_przepisanie_i_kolorowanie();
	ppoz_przepisanie_i_kolorowanie();
	ppop_przepisanie_i_kolorowanie();
}

void CudaTekstury::kolorowanie_oct_t() {
	LARGE_INTEGER countPerSec, tim1, tim2;
	QueryPerformanceFrequency(&countPerSec);
	QueryPerformanceCounter(&tim1);
//	HANDLE_ERROR(hipFree(daneGPU));

//std::future<void> f1;
//std::future<void> f2;
//std::future<void> f3;

std::thread t1;
std::thread t2;
std::thread t3;

	switch (trybWyswietlania)
	{
	case WIZUALIZACJA::TYP_3D:
	
//		f1 = std::async(std::launch::async, [&] {przepisanie_oct_t_bskan();kolorowanieBskan(); });
//		f2 = std::async(std::launch::async, [&] {przepisanie_oct_t_ppop();	kolorowaniePpop(); });
//		f3 = std::async(std::launch::async, [&] {przepisanie_oct_t_ppoz();	kolorowaniePpoz(); });
		
		t1 = std::thread([&] {przepisanie_oct_t_bskan(); kolorowanieBskan(); });
		t2 = std::thread([&] {przepisanie_oct_t_ppop();	kolorowaniePpop(); });
		t3 = std::thread([&] {przepisanie_oct_t_ppoz();	kolorowaniePpoz(); });

		break;
	case WIZUALIZACJA::TYP_2D:
	
//		f1 = std::async(std::launch::async, [&] {przepisanie_oct_t_bskan(); kolorowanieBskan(); });
		t1 = std::thread([&] {przepisanie_oct_t_bskan(); kolorowanieBskan(); });
		break;
	default:
		break;
	}

	
	t1.join();
	if (trybWyswietlania == WIZUALIZACJA::TYP_3D) {
		t2.join();
		t3.join();
	}
	

	QueryPerformanceCounter(&tim2);
	double j = (double)(tim2.QuadPart - tim1.QuadPart) / countPerSec.QuadPart * 1000;
	printf("czas kolorowanie_oct: %f\n", j);
}



void CudaTekstury::kopiowaniePrzekrojow() {

	LARGE_INTEGER countPerSec, tim1, tim2;
	QueryPerformanceFrequency(&countPerSec);
	QueryPerformanceCounter(&tim1);
	hipDeviceSynchronize();

//	std::future<void> f1;
//	std::future<void> f2;
//	std::future<void> f3;

	std::thread t1;
	std::thread t2;
	std::thread t3;

	switch (trybWyswietlania) {

	case WIZUALIZACJA::TYP_3D:
/*	
		f1 = std::async(std::launch::async, [&] {
			for (int i = 0; i < liczbaBskanow; ++i)
				HANDLE_ERROR(hipMemcpy2DToArrayAsync(tabliceCuda[i], 0, 0, daneGPU_bskan_kolor + i*rozmiarAskanu*szerokoscBskanu, szerokoscBskanu * sizeof(uchar4), szerokoscBskanu * sizeof(uchar4), rozmiarAskanu, hipMemcpyDeviceToDevice, streams[0]));

		});

		f2 = std::async(std::launch::async, [&] {

			for (int i = 0; i < liczbaPrzekrojowPoprzecznych; ++i)
				HANDLE_ERROR(hipMemcpy2DToArrayAsync(tabliceCuda[i + liczbaBskanow], 0, 0, daneGPU_ppop_kolor + i*glebokoscPomiaru*szerokoscBskanu, szerokoscBskanu * sizeof(uchar4), szerokoscBskanu * sizeof(uchar4), glebokoscPomiaru, hipMemcpyDeviceToDevice, streams[1]));

		});

		f3 = std::async(std::launch::async, [&] {

			for (int i = 0; i < liczbaPrzekrojowPoziomych; ++i)
				HANDLE_ERROR(hipMemcpy2DToArrayAsync(tabliceCuda[i + liczbaBskanow + liczbaPrzekrojowPoprzecznych], 0, 0, daneGPU_ppoz_kolor + i*glebokoscPomiaru*rozmiarAskanu, glebokoscPomiaru * sizeof(uchar4), glebokoscPomiaru * sizeof(uchar4), rozmiarAskanu, hipMemcpyDeviceToDevice, streams[2]));

		});
*/
	t1 = std::thread([&] {
		for (int i = 0; i < liczbaBskanow; ++i)
			HANDLE_ERROR(hipMemcpy2DToArrayAsync(tabliceCuda[i], 0, 0, daneGPU_bskan_kolor + i*rozmiarAskanu*szerokoscBskanu, szerokoscBskanu * sizeof(uchar4), szerokoscBskanu * sizeof(uchar4), rozmiarAskanu, hipMemcpyDeviceToDevice, streams[0]));

	});

	t2 = std::thread([&] {

		for (int i = 0; i < liczbaPrzekrojowPoprzecznych; ++i)
			HANDLE_ERROR(hipMemcpy2DToArrayAsync(tabliceCuda[i + liczbaBskanow], 0, 0, daneGPU_ppop_kolor + i*glebokoscPomiaru*szerokoscBskanu, szerokoscBskanu * sizeof(uchar4), szerokoscBskanu * sizeof(uchar4), glebokoscPomiaru, hipMemcpyDeviceToDevice, streams[1]));

	});

	t3 = std::thread([&] {
		for (int i = 0; i < liczbaPrzekrojowPoziomych; ++i)
			HANDLE_ERROR(hipMemcpy2DToArrayAsync(tabliceCuda[i + liczbaBskanow + liczbaPrzekrojowPoprzecznych], 0, 0, daneGPU_ppoz_kolor + i*glebokoscPomiaru*rozmiarAskanu, glebokoscPomiaru * sizeof(uchar4), glebokoscPomiaru * sizeof(uchar4), rozmiarAskanu, hipMemcpyDeviceToDevice, streams[2]));

	});
		break;

	case WIZUALIZACJA::TYP_2D:
//		f1 = std::async(std::launch::async, [&] {
//			for (int i = 0; i < liczbaBskanow; ++i)
//				HANDLE_ERROR(hipMemcpy2DToArrayAsync(tabliceCuda[i], 0, 0, daneGPU_bskan_kolor + i*rozmiarAskanu*szerokoscBskanu, szerokoscBskanu * sizeof(uchar4), szerokoscBskanu * sizeof(uchar4), rozmiarAskanu, hipMemcpyDeviceToDevice, streams[0]));

//		});


		t1 = std::thread([&] {
			for (int i = 0; i < liczbaBskanow; ++i)
				HANDLE_ERROR(hipMemcpy2DToArrayAsync(tabliceCuda[i], 0, 0, daneGPU_bskan_kolor + i*rozmiarAskanu*szerokoscBskanu, szerokoscBskanu * sizeof(uchar4), szerokoscBskanu * sizeof(uchar4), rozmiarAskanu, hipMemcpyDeviceToDevice, streams[0]));

		});

		break;
	}
	
	t1.join();
	if (trybWyswietlania == WIZUALIZACJA::TYP_3D) {
		t2.join();
		t3.join();
	}
	;
	
	QueryPerformanceCounter(&tim2);
	double j = (double)(tim2.QuadPart - tim1.QuadPart) / countPerSec.QuadPart * 1000;
	printf("czas kopiowania cudaMemcpy2DtoArray: %f\n", j);
}


__global__ void odswiezaniePrzekroju(uchar4 *dane, const uchar4 *kolory,bool czyRGB) {

	if (threadIdx.x < blockDim.x) {

		unsigned char src = dane[blockIdx.x*blockDim.x + threadIdx.x].w;
		uchar4 tmp;
		if (czyRGB) {

			tmp = kolory[src];
			
		} else {

			tmp.x = kolory[src].w;
			tmp.y = kolory[src].w;
			tmp.z = kolory[src].w;

		}
		
		tmp.w = src;
		dane[blockIdx.x*blockDim.x + threadIdx.x] = tmp;
	}
}

void CudaTekstury::odswiez_bskany() {
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(odswiezaniePrzekroju), hipFuncCachePreferL1);
	dim3 block(szerokoscBskanu);
	dim3 grid(rozmiarAskanu*liczbaBskanow);
	odswiezaniePrzekroju << <grid, block, 0, streams[0] >> > (daneGPU_bskan_kolor, d_mapaKolory_Szarosc,kolor);
	//odswiezaniePrzekroju << <grid, block>> > (daneGPU_bskan_kolor, d_mapaKolory_Szarosc,kolor);
}

void CudaTekstury::odswiez_przekrojePoprzeczne() {
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(odswiezaniePrzekroju), hipFuncCachePreferL1);
	dim3 block(szerokoscBskanu);
	dim3 grid(glebokoscPomiaru*liczbaPrzekrojowPoprzecznych);
	odswiezaniePrzekroju << <grid, block, 0, streams[1] >> > (daneGPU_ppop_kolor, d_mapaKolory_Szarosc,kolor);
	//odswiezaniePrzekroju << <grid, block>> > (daneGPU_ppop_kolor, d_mapaKolory_Szarosc,kolor);
}

void CudaTekstury::odswiez_przekrojePoziome() {
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(odswiezaniePrzekroju), hipFuncCachePreferL1);
	dim3 block(glebokoscPomiaru);
	dim3 grid(rozmiarAskanu*liczbaPrzekrojowPoziomych);
	odswiezaniePrzekroju << < grid, block, 0, streams[2] >> > (daneGPU_ppoz_kolor, d_mapaKolory_Szarosc,kolor);
	//odswiezaniePrzekroju << < grid, block>> > (daneGPU_ppoz_kolor, d_mapaKolory_Szarosc,kolor);
}


void CudaTekstury::pokolorujTeksturyIprzeslijDoTablicCuda() {

	LARGE_INTEGER countPerSec, tim1, tim2;
	QueryPerformanceFrequency(&countPerSec);
	QueryPerformanceCounter(&tim1);

//	std::future<void> f1;
//	std::future<void> f2;
//	std::future<void> f3;

	std::thread t1;
	std::thread t2;
	std::thread t3;

	switch (trybWyswietlania) {

	case WIZUALIZACJA::TYP_3D:
	
//		f1 = std::async(std::launch::async, [&] {odswiez_bskany(); });
//		f2 = std::async(std::launch::async, [&] {odswiez_przekrojePoprzeczne(); });
//		f3 = std::async(std::launch::async, [&] {odswiez_przekrojePoziome(); });
	
		t1 = std::thread([&] {odswiez_bskany(); });
		t2 = std::thread([&] {odswiez_przekrojePoprzeczne(); });
		t3 = std::thread([&] {odswiez_przekrojePoziome(); });
		break;
	case WIZUALIZACJA::TYP_2D:
//		f1 = std::async(std::launch::async, [&] {odswiez_bskany(); });
		t1 = std::thread([&] {odswiez_bskany(); });
		break;
	}

	t1.join();
	if (trybWyswietlania == WIZUALIZACJA::TYP_3D) {
		t2.join();
		t3.join();
	}


	kopiowaniePrzekrojow();
	QueryPerformanceCounter(&tim2);
	double j = (double)(tim2.QuadPart - tim1.QuadPart) / countPerSec.QuadPart * 1000;
	//	printf("czas odswiezania: %f\n", j);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);
	}
}